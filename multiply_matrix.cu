#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include <chrono>
#include <iostream>


#define BLOCK_SIZE 32


//*************************CUDA KERNEL CODE*************************

__global__ void gpu_global_matmul(const double* a, const double* b, double* c, int M, int N, int K){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if(idx < K && idy < M) {
        double sum = 0;
        for(int i = 0; i < N; i++) {
            sum += a[idy * N + i] * b[i * K + idx];
        }
        c[idy * K + idx] = sum;
    }
}


__global__ void gpu_shared_matmul(const double* A, const double* B, double* C, int M, int N, int K){
    int block_row = blockIdx.y;
    int block_col = blockIdx.x;
    
    int row = threadIdx.y;
    int col = threadIdx.x;

    int global_row = block_row * BLOCK_SIZE + row;
    int global_col = block_col * BLOCK_SIZE + col;

    double C_val = 0;

    for(int i = 0; i < ceil((double)N/(double)BLOCK_SIZE); i++){
        __shared__ double A_shared[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double B_shared[BLOCK_SIZE][BLOCK_SIZE];
        
        if(global_row < M && col + i * BLOCK_SIZE < N){
            const double* A_block = &A[N * block_row * BLOCK_SIZE + i * BLOCK_SIZE];
            A_shared[row][col] = A_block[N * row + col];
        }
        else{
            A_shared[row][col] = 0;
        }

        if(row + i * BLOCK_SIZE < N && global_col < K){
            const double* B_block = &B[K * i * BLOCK_SIZE + block_col * BLOCK_SIZE];
            B_shared[row][col] = B_block[K * row + col];
        }
        else{
            B_shared[row][col] = 0;
        }

        __syncthreads();

        for(int j = 0; j < BLOCK_SIZE; j++){
            C_val += A_shared[row][j] * B_shared[j][col];
        }

        __syncthreads();
    }

    if(global_row < M && global_col < K){
        C[global_row * K + global_col] = C_val;
    }
}




//*************************BINDED C++ CODE*************************

namespace py = pybind11;


void cpu_matmul(const py::array_t<double> a, const py::array_t<double> b, py::array_t<double> c, int M, int N, int K){  
    const pybind11::buffer_info h_buff_a = a.request();
    const pybind11::buffer_info h_buff_b = b.request();
    pybind11::buffer_info h_buff_c = c.request();

    const double *h_a, *h_b;
    double *h_c;
    h_a = reinterpret_cast<double*>(h_buff_a.ptr);
    h_b = reinterpret_cast<double*>(h_buff_b.ptr);
    h_c = reinterpret_cast<double*>(h_buff_c.ptr);

    for(int i = 0; i < M; i++){
        for(int j = 0; j < K; j++){
            float sum = 0;
            for(int k = 0; k < N; k++){
                sum += h_a[i*N + k] * h_b[k*K + j];
            }
            h_c[i*K + j] = sum;
        }
    }
}


enum class MEM_TYPE{
    SHARED,
    GLOBAL
};


void gpu_matmul(const py::array_t<const double> a, const py::array_t<const double> b, py::array_t<double> c, int M, int N, int K, MEM_TYPE memory){
    unsigned int sizeOfA = sizeof(double)*M*N;
    unsigned int sizeOfB = sizeof(double)*N*K;
    unsigned int sizeOfC = sizeof(double)*M*K;
    
    const pybind11::buffer_info h_buff_a = a.request();
    const pybind11::buffer_info h_buff_b = b.request();
    pybind11::buffer_info h_buff_c = c.request();

    const double *h_a, *h_b;
    double *h_c;
    h_a = reinterpret_cast<double*>(h_buff_a.ptr);
    h_b = reinterpret_cast<double*>(h_buff_b.ptr);
    h_c = reinterpret_cast<double*>(h_buff_c.ptr);

    hipError_t error;

    double *d_a, *d_b, *d_c;
    error = hipMalloc((void **)&d_a, sizeOfA);
    error = hipMalloc((void **)&d_b, sizeOfB);
    error = hipMalloc((void **)&d_c, sizeOfC);
    
    if (error != hipSuccess) {
        std::cout << "Error in hipMalloc" << std::endl;
        throw std::runtime_error(hipGetErrorString(error));
    }

    error = hipMemcpy(d_a, h_a, sizeOfA, hipMemcpyHostToDevice);
    error = hipMemcpy(d_b, h_b, sizeOfB, hipMemcpyHostToDevice);

    if (error != hipSuccess) {
        std::cout << "Error in first hipMemcpy" << std::endl;
        throw std::runtime_error(hipGetErrorString(error));
    }

    switch(memory){
        case MEM_TYPE::GLOBAL: {
            unsigned int grid_cols = (K + BLOCK_SIZE - 1) / BLOCK_SIZE;
            unsigned int grid_rows = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
            dim3 dim_grid(grid_cols, grid_rows);
            dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
            gpu_global_matmul<<<dim_grid, dim_block>>>(d_a, d_b, d_c, M, N, K);
            break;
        }
        case MEM_TYPE::SHARED: {
            dim3 dim_grid(ceil((double)K / (double)BLOCK_SIZE), ceil((double)M / (double)BLOCK_SIZE));
            dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);
            gpu_shared_matmul<<<dim_grid, dim_block>>>(d_a, d_b, d_c, M, N, K);
            break;
        }
    }


    error = hipMemcpy(h_c, d_c, sizeOfC, hipMemcpyDeviceToHost);

    if (error != hipSuccess) {
        std::cout << "Error in last hipMemcpy" << std::endl;
        throw std::runtime_error(hipGetErrorString(error));
    }

    error = hipFree(d_a);
    error = hipFree(d_b);
    error = hipFree(d_c);

    if (error != hipSuccess) {
        std::cout << "Error in hipFree" << std::endl;
        throw std::runtime_error(hipGetErrorString(error));
    }
}


void global_matmul(const py::array_t<const double> a, const py::array_t<const double> b, py::array_t<double> c, int M, int N, int K){
    gpu_matmul(a, b, c, M, N, K, MEM_TYPE::GLOBAL);
}


void shared_matmul(const py::array_t<const double> a, const py::array_t<const double> b, py::array_t<double> c, int M, int N, int K){
    gpu_matmul(a, b, c, M, N, K, MEM_TYPE::SHARED);
}




//*************************PYBIND11 BINDINGS*************************

PYBIND11_MODULE(gpu_library, m){
    m.doc() = "Plugin for doing GPU accelerated matrix multiply in Python";
    m.def("cuda_global_matrix_multiply", &global_matmul);
    m.def("cuda_shared_matrix_multiply", &shared_matmul);
    m.def("cpu_matrix_multiply", &cpu_matmul);
}